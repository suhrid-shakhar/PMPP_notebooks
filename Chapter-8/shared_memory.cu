#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cstdlib>

#define cudaCheck(call)                                                            \
    {                                                                              \
        hipError_t err = call;                                                    \
        if (err != hipSuccess)                                                    \
        {                                                                          \
            printf("%s at %s: %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
        }                                                                          \
    }


#define MATRIX_WIDTH 1024
#define BLOCK_DIM 8
#define IN_TILE_DIM BLOCK_DIM
#define OUT_TILE_DIM (IN_TILE_DIM - 2)

#define C0 0.1f
#define C1 0.1f
#define C2 0.1f
#define C3 0.1f
#define C4 0.1f
#define C5 0.1f
#define C6 0.1f

__global__ void stencil_kernel(float* in, float* out, unsigned int N) {
    int i = blockIdx.z*OUT_TILE_DIM + threadIdx.z - 1;
    int j = blockIdx.y*OUT_TILE_DIM + threadIdx.y - 1;
    int k = blockIdx.x*OUT_TILE_DIM + threadIdx.x - 1;

    __shared__ float in_s[IN_TILE_DIM][IN_TILE_DIM][IN_TILE_DIM];
    if(i >= 0 && i < N && j >= 0 && j < N && k >= 0 && k < N) {
        in_s[threadIdx.z][threadIdx.y][threadIdx.x] = in[i*N*N + j*N + k];
    }
    __syncthreads();
    if(i >= 1 && i < N - 1 && j >= 1 && j < N - 1 && k >= 1 && k < N - 1) {
         if(threadIdx.z >= 1 && threadIdx.z < IN_TILE_DIM - 1 
            && threadIdx.y >= 1 && threadIdx.y < IN_TILE_DIM - 1 
            && threadIdx.x >= 1 && threadIdx.x < IN_TILE_DIM - 1) 
        {
            out[i*N*N + j*N + k] = C0*in_s[threadIdx.z][threadIdx.y][threadIdx.x]
                                 + C1*in_s[threadIdx.z][threadIdx.y][threadIdx.x - 1]
                                 + C2*in_s[threadIdx.z][threadIdx.y][threadIdx.x + 1]
                                 + C3*in_s[threadIdx.z][threadIdx.y - 1][threadIdx.x]
                                 + C4*in_s[threadIdx.z][threadIdx.y + 1][threadIdx.x]
                                 + C5*in_s[threadIdx.z - 1][threadIdx.y][threadIdx.x]
                                 + C6*in_s[threadIdx.z + 1][threadIdx.y][threadIdx.x];
        }
    }
}
void stencil_cpu(const std::vector<float>& in, std::vector<float>& out, unsigned int N) {
    for (unsigned int i = 1; i < N - 1; ++i) {
        for (unsigned int j = 1; j < N - 1; ++j) {
            for (unsigned int k = 1; k < N - 1; ++k) {
                out[i * N * N + j * N + k] =
                      C0 * in[i * N * N + j * N + k]
                    + C1 * in[i * N * N + j * N + (k - 1)]
                    + C2 * in[i * N * N + j * N + (k + 1)]
                    + C3 * in[i * N * N + (j - 1) * N + k]
                    + C4 * in[i * N * N + (j + 1) * N + k]
                    + C5 * in[(i - 1) * N * N + j * N + k]
                    + C6 * in[(i + 1) * N * N + j * N + k];
            }
        }
    }
}
int main() {
    size_t size = MATRIX_WIDTH * MATRIX_WIDTH * MATRIX_WIDTH * sizeof(float);

    std::vector<float> in_h(MATRIX_WIDTH * MATRIX_WIDTH * MATRIX_WIDTH);
    std::vector<float> out_h(MATRIX_WIDTH * MATRIX_WIDTH * MATRIX_WIDTH, 0);

    for (int i = 0; i < MATRIX_WIDTH * MATRIX_WIDTH * MATRIX_WIDTH; ++i) {
        in_h[i] = 2.0f;
    }

    float *in_d, *out_d;
    cudaCheck(hipMalloc(&in_d, size));
    cudaCheck(hipMalloc(&out_d, size));

    cudaCheck(hipMemcpy(in_d, in_h.data(), size, hipMemcpyHostToDevice));

    dim3 blockDim(BLOCK_DIM, BLOCK_DIM, BLOCK_DIM);
    dim3 gridDim((MATRIX_WIDTH - 2 + OUT_TILE_DIM - 1) / OUT_TILE_DIM,
             (MATRIX_WIDTH - 2 + OUT_TILE_DIM - 1) / OUT_TILE_DIM,
             (MATRIX_WIDTH - 2 + OUT_TILE_DIM - 1) / OUT_TILE_DIM);

    stencil_kernel<<<gridDim, blockDim>>>(in_d, out_d, MATRIX_WIDTH);
    cudaCheck(hipDeviceSynchronize());

    cudaCheck(hipMemcpy(out_h.data(), out_d, size, hipMemcpyDeviceToHost));
    cudaCheck(hipDeviceSynchronize());

    std::vector<float> ref_h(MATRIX_WIDTH * MATRIX_WIDTH * MATRIX_WIDTH, 0);
    stencil_cpu(in_h, ref_h, MATRIX_WIDTH);

    // Compare results
    int mismatch = 0;
    for (int i = 1; i < MATRIX_WIDTH - 1 && !mismatch; ++i) {
        for (int j = 1; j < MATRIX_WIDTH - 1 && !mismatch; ++j) {
            for (int k = 1; k < MATRIX_WIDTH - 1 && !mismatch; ++k) {
                int idx = i * MATRIX_WIDTH * MATRIX_WIDTH + j * MATRIX_WIDTH + k;
                float diff = fabs(ref_h[idx] - out_h[idx]);
                if (diff > 1e-5f) {
                    std::cout << "Mismatch at (" << i << "," << j << "," << k << ") "
                            << "CPU: " << ref_h[idx] << ",GPU: "<< out_h[idx] << std::endl;               
                    mismatch++;
                }
            }
        }
    }
    if(!mismatch) 
        std::cout<<"CPU Matches GPU result."<<std::endl;
    else 
        std::cout<<"Result incorrect."<<std::endl;
    hipFree(in_d);
    hipFree(out_d);

    return 0;
}