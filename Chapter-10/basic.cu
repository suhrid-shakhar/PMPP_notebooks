#include <stdio.h>
#include <hip/hip_runtime.h>

#define cudaCheck(call)                                                            \
    {                                                                              \
        hipError_t err = call;                                                    \
        if (err != hipSuccess)                                                    \
        {                                                                          \
            printf("%s at %s: %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
        }                                                                          \
    }

#define LENGTH 1024
#define BLOCK_SIZE 512

__global__ void parallelReductionKernel(float *input, float *output)
{
    unsigned int i = threadIdx.x * 2;
    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2)
    {
        if(threadIdx.x % stride == 0)
        {
            input[i] += input [i+stride];
        }
        __syncthreads();
    }
    if(threadIdx.x == 0)
        *output = input[0];
}
__global__ void initData(float *data)
{
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;
    while (idx < LENGTH) 
    {
        data[idx] = 2.5f;
        idx += stride;
    }
}

void cpuReduction(float *input, float *output)
{
    float sum = 0.0f;
    for(int i=0; i<LENGTH; i++)
        sum += input[i];
    *output = sum;
}

int main(int argc, char* argv[])
{
    float *data_h, *data_d, *result_h, *result_d, result_cpu;
    int size = sizeof(float) * LENGTH;
    hipStream_t memcpy;
    cudaCheck(hipStreamCreateWithFlags(&memcpy, hipStreamNonBlocking));

    cudaCheck(hipMalloc(&data_d, size));
    cudaCheck(hipHostMalloc(&data_h, size, hipHostMallocDefault));

    initData<<<(LENGTH + BLOCK_SIZE -1)/BLOCK_SIZE, BLOCK_SIZE>>>(data_d);
    cudaCheck(hipDeviceSynchronize());
    cudaCheck(hipMemcpyAsync(data_h, data_d, size, hipMemcpyDeviceToHost, memcpy));

    cudaCheck(hipMalloc(&result_d, sizeof(float)));
    cudaCheck(hipHostMalloc(&result_h, sizeof(float), hipHostMallocDefault));

    dim3 block(BLOCK_SIZE);
    // int gridSize = (LENGTH + block.x - 1)/ block.x;
    dim3 grid(1);
    parallelReductionKernel<<<grid, block>>>(data_d, result_d);
    cudaCheck(hipDeviceSynchronize());
    cpuReduction(data_h, &result_cpu);
    cudaCheck(hipMemcpy(result_h, result_d, sizeof(float), hipMemcpyDeviceToHost));

    if(result_cpu == *result_h) 
        printf("GPU value matches CPU value\n");
    else 
        printf("CPU and GPU value do not match. CPU = %f, GPU = %f.\n", result_cpu, *result_h);
    
    cudaCheck(hipFree(data_d));
    cudaCheck(hipFree(result_d));
    cudaCheck(hipHostFree(data_h));
    cudaCheck(hipHostFree(result_h));

    return 0;
}